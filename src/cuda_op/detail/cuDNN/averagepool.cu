#include "hip/hip_runtime.h"
#include "cuda_op/detail/cuDNN/averagepool2D.hpp"
#include <hip/hip_runtime.h>
#include <glog/logging.h>

namespace cu_op_mem {

template <typename T>
__global__ void averagepool2D_kernel(const T* input, T* output, int input_height, int input_width, int output_height,
                                     int output_width, int pool_height, int pool_width, int stride_height,
                                     int stride_width) {
    // 共享内存块大小配置 (32x32)
    constexpr int TILE_DIM   = 32;
    constexpr int BLOCK_ROWS = 8;
    __shared__ T shared_block[TILE_DIM][TILE_DIM];

    const int output_x = blockIdx.x * TILE_DIM + threadIdx.x;
    const int output_y = blockIdx.y * TILE_DIM + threadIdx.y;

    if (output_x >= output_width || output_y >= output_height) return;

    const int input_x_start = output_x * stride_width;
    const int input_y_start = output_y * stride_height;
    const int input_x_end = min(input_x_start + pool_width, input_width);
    const int input_y_end = min(input_y_start + pool_height, input_height);

    T sum = 0;
    int count = 0;
    for (int y = input_y_start; y < input_y_end; y += BLOCK_ROWS) {
        for (int x = input_x_start; x < input_x_end; x += TILE_DIM) {
            const int load_x = x + threadIdx.x;
            const int load_y = y + threadIdx.y;
            if (load_x < input_width && load_y < input_height) {
                shared_block[threadIdx.y][threadIdx.x] = input[load_y * input_width + load_x];
            } else {
                shared_block[threadIdx.y][threadIdx.x] = 0;
            }
            __syncthreads();
            const int search_height = min(BLOCK_ROWS, input_y_end - y);
            const int search_width  = min(TILE_DIM, input_x_end - x);
            for (int i = 0; i < search_height; ++i) {
                for (int j = 0; j < search_width; ++j) {
                    sum += shared_block[i][j];
                    ++count;
                }
            }
            __syncthreads();
        }
    }
    output[output_y * output_width + output_x] = sum / (T)count;
}

template <typename T>
StatusCode AveragePool2D<T>::Forward(const Tensor<T>& input, Tensor<T>& output, int dim_h, int dim_w) {
    const auto& input_shape = input.shape();
    if (input_shape.size() == 2) {
        // 原二维实现
        int input_height = input_shape[0];
        int input_width = input_shape[1];
        int output_height = (input_height - pool_height_) / stride_height_ + 1;
        int output_width = (input_width - pool_width_) / stride_width_ + 1;
        std::vector<std::size_t> output_shape = {static_cast<std::size_t>(output_height), static_cast<std::size_t>(output_width)};
        output.resize(output_shape);
        dim3 block_size(16, 16);
        dim3 grid_size((output_width + block_size.x - 1) / block_size.x, (output_height + block_size.y - 1) / block_size.y);
        averagepool2D_kernel<T><<<grid_size, block_size>>>(input.data(), output.data(), input_height, input_width, output_height, output_width, pool_height_, pool_width_, stride_height_, stride_width_);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            LOG(ERROR) << "AveragePool2D kernel failed: " << hipGetErrorString(err);
            return StatusCode::CUDA_ERROR;
        }
        hipDeviceSynchronize();
        return StatusCode::SUCCESS;
    } else if (input_shape.size() == 4) {
        // 四维张量 [N, C, H, W]
        int N = input_shape[0];
        int C = input_shape[1];
        int H = input_shape[2];
        int W = input_shape[3];
        int output_height = (H - pool_height_) / stride_height_ + 1;
        int output_width = (W - pool_width_) / stride_width_ + 1;
        std::vector<std::size_t> output_shape = {static_cast<std::size_t>(N), static_cast<std::size_t>(C), static_cast<std::size_t>(output_height), static_cast<std::size_t>(output_width)};
        output.resize(output_shape);
        int batch = N * C;
        const T* input_ptr = input.data();
        T* output_ptr = output.data();
        for (int i = 0; i < batch; ++i) {
            const T* in = input_ptr + i * H * W;
            T* out = output_ptr + i * output_height * output_width;
            dim3 block_size(16, 16);
            dim3 grid_size((output_width + block_size.x - 1) / block_size.x, (output_height + block_size.y - 1) / block_size.y);
            averagepool2D_kernel<T><<<grid_size, block_size>>>(in, out, H, W, output_height, output_width, pool_height_, pool_width_, stride_height_, stride_width_);
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            LOG(ERROR) << "AveragePool2D kernel failed: " << hipGetErrorString(err);
            return StatusCode::CUDA_ERROR;
        }
        hipDeviceSynchronize();
        return StatusCode::SUCCESS;
    } else {
        LOG(ERROR) << "AveragePool2D only supports 2D or 4D input, got " << input_shape.size() << "D";
        return StatusCode::TENSOR_DIMONSION_MISMATCH;
    }
}

template class AveragePool2D<float>;
template class AveragePool2D<double>;

} // namespace cu_op_mem
