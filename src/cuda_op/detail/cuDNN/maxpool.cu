#include "hip/hip_runtime.h"
#include "cuda_op/detail/cuDNN/maxpool.hpp"
#include <hip/hip_runtime.h>
#include <glog/logging.h>

namespace cu_op_mem {

// 优化的MaxPool2D kernel，改进共享内存使用和边界处理
template <typename T>
__global__ void maxpool2D_optimized_kernel(const T* input, T* output, int input_height, int input_width, int output_height,
                                          int output_width, int pool_height, int pool_width, int stride_height,
                                          int stride_width) {
    // 优化的块大小配置
    constexpr int TILE_DIM = 16;  // 减小块大小以减少共享内存使用

    // 共享内存缓存输入块
    __shared__ T shared_block[TILE_DIM][TILE_DIM];

    // 计算输出坐标
    const int output_x = blockIdx.x * TILE_DIM + threadIdx.x;
    const int output_y = blockIdx.y * TILE_DIM + threadIdx.y;

    // 边界检查
    if (output_x >= output_width || output_y >= output_height) return;

    // 计算输入区域起始坐标
    const int input_x_start = output_x * stride_width;
    const int input_y_start = output_y * stride_height;

    // 计算输入区域结束坐标 (考虑边界)
    const int input_x_end = min(input_x_start + pool_width, input_width);
    const int input_y_end = min(input_y_start + pool_height, input_height);

    // 使用向量化内存访问优化
    T max_val = -INFINITY;
    
    // 直接计算最大值，避免不必要的共享内存操作
    for (int y = input_y_start; y < input_y_end; ++y) {
        for (int x = input_x_start; x < input_x_end; ++x) {
            if (x < input_width && y < input_height) {
                T val = input[y * input_width + x];
                max_val = max(max_val, val);
            }
        }
    }

    // 写入输出
    output[output_y * output_width + output_x] = max_val;
}

// 使用共享内存的MaxPool2D kernel（用于大池化窗口）
template <typename T>
__global__ void maxpool2D_shared_kernel(const T* input, T* output, int input_height, int input_width, int output_height,
                                       int output_width, int pool_height, int pool_width, int stride_height,
                                       int stride_width) {
    constexpr int TILE_DIM = 32;
    constexpr int BLOCK_ROWS = 8;

    // 共享内存缓存输入块
    __shared__ T shared_block[TILE_DIM][TILE_DIM];

    // 计算输出坐标
    const int output_x = blockIdx.x * TILE_DIM + threadIdx.x;
    const int output_y = blockIdx.y * TILE_DIM + threadIdx.y;

    // 边界检查
    if (output_x >= output_width || output_y >= output_height) return;

    // 计算输入区域起始坐标
    const int input_x_start = output_x * stride_width;
    const int input_y_start = output_y * stride_height;

    // 计算输入区域结束坐标 (考虑边界)
    const int input_x_end = min(input_x_start + pool_width, input_width);
    const int input_y_end = min(input_y_start + pool_height, input_height);

    // 每个线程处理多个输入元素 (优化内存访问)
    T max_val = -INFINITY;
    for (int y = input_y_start; y < input_y_end; y += BLOCK_ROWS) {
        for (int x = input_x_start; x < input_x_end; x += TILE_DIM) {
            // 将输入块加载到共享内存 (协作加载)
            const int load_x = x + threadIdx.x;
            const int load_y = y + threadIdx.y;

            if (load_x < input_width && load_y < input_height) {
                shared_block[threadIdx.y][threadIdx.x] = input[load_y * input_width + load_x];
            } else {
                shared_block[threadIdx.y][threadIdx.x] = -INFINITY;
            }
            __syncthreads();

            // 在共享内存块中查找最大值
            const int search_height = min(BLOCK_ROWS, input_y_end - y);
            const int search_width = min(TILE_DIM, input_x_end - x);

            for (int i = 0; i < search_height; ++i) {
                for (int j = 0; j < search_width; ++j) {
                    max_val = max(max_val, shared_block[i][j]);
                }
            }
            __syncthreads();
        }
    }

    // 写入输出
    output[output_y * output_width + output_x] = max_val;
}

template <typename T>
StatusCode MaxPool2D<T>::Forward(const Tensor<T>& input, Tensor<T>& output, int dim_h, int dim_w) {
    const auto& input_shape = input.shape();
    if (input_shape.size() == 2) {
        // 原二维实现
        int input_height = input_shape[0];
        int input_width = input_shape[1];
        int output_height = (input_height - pool_height_) / stride_height_ + 1;
        int output_width = (input_width - pool_width_) / stride_width_ + 1;
        std::vector<std::size_t> output_shape = {static_cast<std::size_t>(output_height), static_cast<std::size_t>(output_width)};
        output.resize(output_shape);
        // 根据池化窗口大小选择合适的kernel
        if (pool_height_ * pool_width_ > 16) {
            // 大池化窗口使用共享内存kernel
            dim3 block_size(16, 16);
            dim3 grid_size((output_width + block_size.x - 1) / block_size.x, (output_height + block_size.y - 1) / block_size.y);
            maxpool2D_shared_kernel<T><<<grid_size, block_size>>>(input.data(), output.data(), input_height, input_width, output_height, output_width, pool_height_, pool_width_, stride_height_, stride_width_);
        } else {
            // 小池化窗口使用优化kernel
            dim3 block_size(16, 16);
            dim3 grid_size((output_width + block_size.x - 1) / block_size.x, (output_height + block_size.y - 1) / block_size.y);
            maxpool2D_optimized_kernel<T><<<grid_size, block_size>>>(input.data(), output.data(), input_height, input_width, output_height, output_width, pool_height_, pool_width_, stride_height_, stride_width_);
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            LOG(ERROR) << "MaxPool2D kernel failed: " << hipGetErrorString(err);
            return StatusCode::CUDA_ERROR;
        }
        hipDeviceSynchronize();
        return StatusCode::SUCCESS;
    } else if (input_shape.size() == 4) {
        // 四维张量 [N, C, H, W]
        int N = input_shape[0];
        int C = input_shape[1];
        int H = input_shape[2];
        int W = input_shape[3];
        int output_height = (H - pool_height_) / stride_height_ + 1;
        int output_width = (W - pool_width_) / stride_width_ + 1;
        std::vector<std::size_t> output_shape = {static_cast<std::size_t>(N), static_cast<std::size_t>(C), static_cast<std::size_t>(output_height), static_cast<std::size_t>(output_width)};
        output.resize(output_shape);
        int batch = N * C;
        const T* input_ptr = input.data();
        T* output_ptr = output.data();
        for (int i = 0; i < batch; ++i) {
            const T* in = input_ptr + i * H * W;
            T* out = output_ptr + i * output_height * output_width;
            
            // 根据池化窗口大小选择合适的kernel
            if (pool_height_ * pool_width_ > 16) {
                dim3 block_size(16, 16);
                dim3 grid_size((output_width + block_size.x - 1) / block_size.x, (output_height + block_size.y - 1) / block_size.y);
                maxpool2D_shared_kernel<T><<<grid_size, block_size>>>(in, out, H, W, output_height, output_width, pool_height_, pool_width_, stride_height_, stride_width_);
            } else {
                dim3 block_size(16, 16);
                dim3 grid_size((output_width + block_size.x - 1) / block_size.x, (output_height + block_size.y - 1) / block_size.y);
                maxpool2D_optimized_kernel<T><<<grid_size, block_size>>>(in, out, H, W, output_height, output_width, pool_height_, pool_width_, stride_height_, stride_width_);
            }
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            LOG(ERROR) << "MaxPool2D kernel failed: " << hipGetErrorString(err);
            return StatusCode::CUDA_ERROR;
        }
        hipDeviceSynchronize();
        return StatusCode::SUCCESS;
    } else {
        LOG(ERROR) << "MaxPool2D only supports 2D or 4D input, got " << input_shape.size() << "D";
        return StatusCode::TENSOR_DIMONSION_MISMATCH;
    }
}

template class MaxPool2D<float>;
template class MaxPool2D<double>;

} // namespace cu_op_mem